#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include "statement.h"

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

__global__ void storecheckpoints
		(
			int flag,int nc,int nxx,int nzz,
			float *d_p,float *d_vx,float *d_vz,
			float *d_fipx,float *d_fipz,
			float *d_fivxx,float *d_fivzz,
			float *store_p,float *store_vx,float *store_vz,
			float *store_fipx,float *store_fipz,
			float *store_fivxx,float *store_fivzz
		);
		
__global__ void addsource(int nx,int nz,int xs,int zs,float *check_d_ps,float *d_rik,float dt);

__global__ void initialize_kernel(
					int nxx,int nzz,int pml,
					float *d_p,float *d_vx,float *d_vz,
					float *d_fipx,float *d_fipz,
					float *d_fivxx,float *d_fivzz,
					float *d_img,float *d_ssg,
					float *d_pb,float *d_vxb,float *d_vzb,
					float *d_fipxb,float *d_fipzb,
					float *d_fivxxb,float *d_fivzzb
				);
				
__global__ void initialize_kernel_ins(
					int nxx,int nzz,int pml,
					float *d_p,float *d_vx,float *d_vz,
					float *d_fipx,float *d_fipz,
					float *d_fivxx,float *d_fivzz,float *d_aux
				);
				
                                  
__global__ void fuzhi_kernel(
					int nxx,int nzz,float *d_vx,float *d_vz,
					hipfftComplex *d_dis0,hipfftComplex *d_dis1
			    );
			    
			    
__global__ void fuzhi_kernel_p (int nxx,int nzz,float *d_p,hipfftComplex *d_dis0);

__global__ void multt_kernel(float eta,float dt,int nt,float tao,
			     int NXZ,int nxx,int nzz,
			     float *k2da,hipfftComplex *d_dis0
			     );

__global__ void fuzhi_kernel_inverse(int nxx,int nzz,float *d_aux,hipfftComplex *d_dis0);

__global__ void shift_vxz_kernel(
					float *kxsmr,float *kxsmi,float *kzscr,float *kzsci,
					hipfftComplex *d_outdatax,hipfftComplex *d_outdataz,
					hipfftComplex *d_indatax,hipfftComplex *d_indataz,
					int nxx,int nzz
				);
				
__global__ void shift_pxz_kernel(
					float *kxscr,float *kxsci,float *kzsmr,float *kzsmi,
     					hipfftComplex *d_dis0,
     					hipfftComplex *d_dis1,hipfftComplex *d_dis2,
     					int nxx,int nzz
     				);

__global__ void multv_kernel(int N1,int nxx,int nzz,int ig,float *d_V,hipfftComplex *d_psk,hipfftComplex *d_pskf);

__global__ void fuzhi_to_ifftrow(int ig,int nxx,int nzz,int N1,hipfftComplex *d_pskf,hipfftComplex *d_pifft);

__global__ void fuzhi_r_kernel(int nxx,int nzz,int jg,int N1,double *d_pst);

__global__ void sumlw_kernel(int nxx,int nzz,int jg,int N1,int row,int ig,double *d_pst,double *d_A,hipfftComplex *d_pifft);

__global__ void sumlw2_kernel(int nxx,int nzz,int N1,int col,double *d_pst,float *d_U,float *d_psc);

__global__ void update_vxx_vzz(
       				int nxx,int nzz,float *d_fi_vx,float *d_fi_vz,
       				float *di_a_x,float *di_b_x,float *di_a_z,float *di_b_z,
       				hipfftComplex *d_outdatax,hipfftComplex *d_outdataz
       			       );

__global__ void addsa_kernel(
				int restart,int pml,int nxx,int nzz,
				float *d_vp,float *d_rho,float dt,
				float *d_p,hipfftComplex *d_dis0,hipfftComplex *d_dis3,
				int xs,int zs,int nt,float *d_Rik,float *d_ssg
			    );
			    
__global__ void addsa_b_kernel(
				int pml,int nxx,int nzz,
				float *d_vp,float *d_rho,float dt,
				float *d_p,hipfftComplex *d_dis0,hipfftComplex *d_dis3,
				int nt,int tmax,float *d_obs,
				int *rindex, int *flagrec,int nr
			    );
			    
__global__ void vx_cal_kernel(
     				int nxx,int nzz,float dt,float *d_bbx,
     				float *d_fi_px,float *dh_a_x,
     				float *dh_b_x,hipfftComplex *d_outdatax,float *d_vx
     			     );

__global__ void vz_cal_kernel(
     				int nxx,int nzz,float dt,float *d_bbz,
     				float *d_fi_pz,float *dh_a_z,
     				float *dh_b_z,hipfftComplex *d_outdataz,float *d_vz
     			     );
     			     
__global__ void rtm_image(
				int nx,int nz,int nxx,int nzz,
				float *d_p,float *d_pb,
				float *d_img,float *d_imgf
			 );
			 
__global__ void filter_image(
				int nx,int nz,float dx,float dz,
				float *d_img,float *d_imgf
			    );

__global__ void sumpxpz(
       				int nxx,int nzz,
       				hipfftComplex *d_outdatax,hipfftComplex *d_outdataz,
				hipfftComplex *d_dis0
       			       );

__global__ void  sum_total(int nxx,int nzz,
			   hipfftComplex *d_dis0,hipfftComplex *d_dis2,
			   hipfftComplex *d_dis3,hipfftComplex *d_dis4,
			   float *d_total,float *d_vp,float *d_lam,
			   float *d_gam,float *d_vwd,float dt);

__global__ void multtt_kernel(int nxx,int nzz,float *d_k2d3,hipfftComplex *d_dis0,hipfftComplex *d_dis1);
     	
extern "C" float get_max_tao(int nxx,int nzz,float *tao);

extern "C" float get_min_tao(int nxx,int nzz,float *tao);		       
                         
extern "C" void  expand_pml_areas(int nx,int nz,int nxx,int nzz,int pml,float *vp);

extern "C" void  select_part_model(int,float mlx,float dx,int nxt,int nzt,int nx,int nz,int nxx,int nzz,float *vpt,float *vp);

extern "C" void input_geophone_pos(int shotid,int nr,float *gxp,char fname[]);

extern "C" void acsglowrank2d(int row,int col,int NXZ,int Nh,
			       float *vp,float *tao,float *lam,float *gam,
			       float *k2dh,float dt,float *U,float *V,
			       double *A);
			       
extern "C" void checka_error(int row,int col,int N1,int N2,float *U,double *A,float *V,
			     float vp,float gam,float tao,float dt,
                             float *k2dh,float *We,float *Wa,int is);
                             
extern "C" void consist_Vt(int row,int nxx,int nzz,int nxh,int nzh,int N1,int N2,float *V,float *Vt);


extern "C" void projectomodel(
				int pml,float mlx,float dx,int nxt,int nzt,
			      	int nx,int nz,int nxx,int nzz,float *vpt,float *vp
			     );
			     
extern "C" void removedirect(
				int nrmx,int tmax,int nr,float dt,float fd,float xs,float zs,
				float *gxp,float *gzp,float v0,float *recsim
			    );


void Alloc_host_device_mem(
				int num,int nrmx,int nx,int nz,
				int nxx,int nzz,int nxh,int nzh,int NXZ,int Nh,
			   	int tmax,int GPU_N,struct MultiGPU singpu[]
			  );
			   
extern "C" enum action revolve(int* check,int* capo,int* fine,int snaps,int* info);



extern "C" void acforward(
				int snapnum,int GPU_N,int tmax,float dt,float *rik,int myid,int numprocs,
			  	int nshot,struct MultiGPU singpu[],float *gxp,float *gzp,
			  	int nrmx,int nx,int nz,int nxx,int nzz,
		          	int pml,float dx,float dz,float *k2da,float *k2d1,float *k2d2,float *k2d3,
		          	float sroffmx,float disx,
		          	float *vpt,
		          	float *rhot,float *bbxt,float *bbzt,
			  	int nxt,int nzt,int *tracenum,
			  	float *i_ax,float *i_bx,float *i_az,float *i_bz,
			 	float *h_ax,float *h_bx,float *h_az,float *h_bz,
			  	float *kxsmr,float *kxscr,float *kzsmr,float *kzscr,
			  	float *kxsmi,float *kxsci,float *kzsmi,float *kzsci,
			  	float *img_sp,float *ssg_sp,float fobs,float v0,
                          	char fname4[40],char fname5[40],char recname[60]
                          )                                
{
    // GPU configuration//
    dim3 dimBlock(BLOCK_WIDTH,BLOCK_HEIGHT);
    dim3 dimGrid((nxx+dimBlock.x-1)/dimBlock.x,(nzz+dimBlock.y-1)/dimBlock.y);

    
    int i,j,t,is,ns,nt;
    int ip;
    int shotid;
    int NXZ=nxx*nzz;
    
    int nxh=(int)(nxx/2)+1;
    int nzh=(int)(nzz/2)+1;
    int Nh=nxh*nzh;

    //printf("%d\n",snapnum);

    // allocate device memory for variables //
    Alloc_host_device_mem(snapnum,nrmx,nx,nz,nxx,nzz,nxh,nzh,NXZ,Nh,tmax,GPU_N,singpu);
    
    // create multi-stream //
    Multistream plansm[GPU_N];
    for(i=0;i<GPU_N;i++)
    {
	hipSetDevice(i);
	hipStreamCreate(&plansm[i].stream);	
	hipfftSetStream(singpu[i].plan2dforward,plansm[i].stream);
	hipfftSetStream(singpu[i].plan2dinverse,plansm[i].stream);
    }

     
    // copy values from host to device //
    for(i=0;i<GPU_N;i++)
    {
	hipSetDevice(i); 
	hipMemcpyAsync(singpu[i].d_rik,rik,tmax*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);

	
	hipMemcpyAsync(singpu[i].d_k2d1,k2d1,NXZ*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	

	hipMemcpyAsync(singpu[i].d_i_az,i_az,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_i_bz,i_bz,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
        hipMemcpyAsync(singpu[i].d_i_ax,i_ax,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_i_bx,i_bx,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	
	hipMemcpyAsync(singpu[i].d_h_az,h_az,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_h_bz,h_bz,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
        hipMemcpyAsync(singpu[i].d_h_ax,h_ax,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_h_bx,h_bx,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	
			  
	hipMemcpyAsync(singpu[i].d_kxsmr,kxsmr,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_kxscr,kxscr,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_kxsmi,kxsmi,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_kxsci,kxsci,nxx*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	
	hipMemcpyAsync(singpu[i].d_kzsmr,kzsmr,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_kzscr,kzscr,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_kzsmi,kzsmi,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
	hipMemcpyAsync(singpu[i].d_kzsci,kzsci,nzz*sizeof(float),hipMemcpyHostToDevice,plansm[i].stream);
    }

    //printf("%f\n",rik[0]);

    
    float xs,zs;


    //char snapname[60];
    char obsname[80];
    FILE *fr;
    //FILE *fs=fopen("snap0.bin","wb");
    //FILE *fss=fopen("snap00.bin","wb");
    //FILE *fb=fopen("snap1.bin","wb");
    
    //int ick;
    
   // char checkname[60];
   // FILE *fck;
    
    //int flags;
    
    // for wavefield reconstruction //
    int check, capo, fine, steps, snaps, info;
    enum action whatodo;
    capo = 0;
    snaps = snapnum;
    steps = tmax ;
    info  = 3 ;
    fine  = steps + capo;
    check = -1;
    
    int ts = 0;
    int te = tmax-1;
    int tea = 0 ;
    int restart = 0;
	
   //FILE *fv=fopen("data.bin","wb");
	


    for(ns=0;ns<nshot;ns++)   // shot iteration //
    {
    	//flags = 1;
    	restart = 0; ts =0; te = tmax-1; tea = 0 ;
	capo = 0; snaps = snapnum; steps = tmax; info  = 3; fine  = steps + capo; check = -1;
    	

	//---------------------------------------------------------------------------//
	// 1. get velocity for each gpu //
	// 2. get source x and z grid index for each gpu //
	// 3. get receiver x and z grid index for each gpu with the trace number nr //
        
	for(ip=0;ip<GPU_N;ip++)       // gpu iteration  //
	{
		shotid=singpu[ip].shotid[ns];
		xs=singpu[ip].sxp[ns];
		zs=singpu[ip].szp[ns];
		printf("myid=%d shotid=%d nx=%d nz=%d xs=%f zs=%f\n",myid,shotid,nx,nz,xs,zs);
		
		singpu[ip].lgest=10000;
		singpu[ip].rgest=-1;

		// input vp from vpt model //
		singpu[ip].mlx=xs-sroffmx-disx;

		select_part_model(pml,singpu[ip].mlx,dx,nxt,nzt,nx,nz,nxx,nzz,vpt,singpu[ip].h_vp);
               
                
                select_part_model(pml,singpu[ip].mlx,dx,nxt,nzt,nx,nz,nxx,nzz,rhot,singpu[ip].h_rho);
                select_part_model(pml,singpu[ip].mlx,dx,nxt,nzt,nx,nz,nxx,nzz,bbxt,singpu[ip].h_bbx);
                select_part_model(pml,singpu[ip].mlx,dx,nxt,nzt,nx,nz,nxx,nzz,bbzt,singpu[ip].h_bbz);
                
                expand_pml_areas(nx,nz,nxx,nzz,pml,singpu[ip].h_vp);
		
                
                expand_pml_areas(nx,nz,nxx,nzz,pml,singpu[ip].h_rho);
                expand_pml_areas(nx,nz,nxx,nzz,pml,singpu[ip].h_bbx);
                expand_pml_areas(nx,nz,nxx,nzz,pml,singpu[ip].h_bbz);
                
                

		//fwrite(singpu[ip].h_vp,sizeof(float),NXZ,fv);
                //fwrite(singpu[ip].h_vwd,sizeof(float),NXZ,fv);
                //fwrite(singpu[ip].h_tao,sizeof(float),NXZ,fv);
		//fwrite(singpu[ip].h_gam,sizeof(float),NXZ,fv);
		//fwrite(singpu[ip].h_lam,sizeof(float),NXZ,fv);
		//fwrite(singpu[ip].h_rho,sizeof(float),NXZ,fv);
		//fwrite(singpu[ip].h_bbx,sizeof(float),NXZ,fv);
		//fwrite(singpu[ip].h_bbz,sizeof(float),NXZ,fv);
		
		 // locate source grid index on the model //
                singpu[ip].zsg=(int)(zs/dz+0.5)+pml;
                singpu[ip].xsg=(int)((sroffmx+disx)/dx+0.5)+pml;
		if(singpu[ip].xsg<0||singpu[ip].xsg>=nxx||singpu[ip].zsg<0||singpu[ip].zsg>=nzz)
                {
			printf("%d shot grid index exceeds model!\n",shotid);
			exit(0);
		}
		
		// locate receiver's grid index on the model //
                singpu[ip].nr=tracenum[shotid];
                input_geophone_pos(shotid,singpu[ip].nr,gxp,fname4);
	        input_geophone_pos(shotid,singpu[ip].nr,gzp,fname5);
	        
	        for(i=0;i<NXZ;i++)
		{
			singpu[ip].flagrec[i]=0;
		}


                for(i=0;i<singpu[ip].nr;i++)
		{
		    singpu[ip].gzg[i]=(int)(gzp[i]/dz+0.5)+pml;
		    if(gxp[i]>=xs){singpu[ip].gxg[i]=singpu[ip].xsg+(int)((gxp[i]-xs)/dx+0.5);}
		    else{singpu[ip].gxg[i]=singpu[ip].xsg-(int)((xs-gxp[i])/dx+0.5);}

		    if(singpu[ip].gxg[i]<pml){printf("warning: receiver x grid index in left pml area!\n");singpu[ip].gxg[i]=pml;}
                    if(singpu[ip].gxg[i]>=nxx-pml){printf("warning: receiver x grid index in right pml area!\n");singpu[ip].gxg[i]=nxx-pml-1;}
                    
                    // mark receiver grid index //
                    is=singpu[ip].gxg[i]*nzz+singpu[ip].gzg[i];
                    singpu[ip].flagrec[is]=1;
                    singpu[ip].rindex[is]=i;
                    
                    if(singpu[ip].gxg[i]>singpu[ip].rgest){singpu[ip].rgest=singpu[ip].gxg[i];}
		    if(singpu[ip].gxg[i]<singpu[ip].lgest){singpu[ip].lgest=singpu[ip].gxg[i];}
                }
                
                for(i=0;i<nxt*nzt;i++)
		{	
			singpu[ip].h_img_pj[i]=0.0; singpu[ip].h_ssg_pj[i]=0.0;
		}
		
		
		// input observed data //
		strcpy(obsname,"");
                sprintf(obsname,"%s%d_%d.bin",recname,shotid,singpu[ip].nr); 
                fr=fopen(obsname,"rb");
		fread(&singpu[ip].h_obs[0],sizeof(float),tmax*singpu[ip].nr,fr);
		fclose(fr);
		
		
		
		removedirect(nrmx,tmax,singpu[ip].nr,dt,fobs,xs,zs,gxp,gzp,v0,singpu[ip].h_obs);
		
		

	} // end gpu iteration 
        //---------------------------------------------------------------------------//


	// give a zero initial value //
	for(ip=0;ip<GPU_N;ip++)
	{
		hipSetDevice(ip);
		
		hipMemcpyAsync(singpu[ip].d_vp,singpu[ip].h_vp,NXZ*sizeof(float),hipMemcpyHostToDevice,plansm[ip].stream);
		
		hipMemcpyAsync(singpu[ip].d_rho,singpu[ip].h_rho,NXZ*sizeof(float),hipMemcpyHostToDevice,plansm[ip].stream);
		hipMemcpyAsync(singpu[ip].d_bbx,singpu[ip].h_bbx,NXZ*sizeof(float),hipMemcpyHostToDevice,plansm[ip].stream);
		hipMemcpyAsync(singpu[ip].d_bbz,singpu[ip].h_bbz,NXZ*sizeof(float),hipMemcpyHostToDevice,plansm[ip].stream);
		
		
		
		hipMemcpyAsync(singpu[ip].d_flagrec,singpu[ip].flagrec,nxx*nzz*sizeof(int),hipMemcpyHostToDevice,plansm[ip].stream);
		hipMemcpyAsync(singpu[ip].d_rindex,singpu[ip].rindex,nxx*nzz*sizeof(int),hipMemcpyHostToDevice,plansm[ip].stream);
		
		initialize_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				(
					nxx,nzz,pml,
					singpu[ip].d_p,singpu[ip].d_vx,singpu[ip].d_vz,
				 	singpu[ip].d_fipx,singpu[ip].d_fipz,
				 	singpu[ip].d_fivxx,singpu[ip].d_fivzz,
				 	singpu[ip].d_img,singpu[ip].d_ssg,
				 	singpu[ip].d_pb,singpu[ip].d_vxb,singpu[ip].d_vzb,
				 	singpu[ip].d_fipxb,singpu[ip].d_fipzb,
				 	singpu[ip].d_fivxxb,singpu[ip].d_fivzzb
				);
	}

	//for(ip=0;ip<GPU_N;ip++)
	//{
	//	hipSetDevice(ip);
		hipDeviceSynchronize();
	//}
	
	do
	{
		whatodo = revolve(&check, &capo, &fine, snaps, &info);
		
		// case 1: store checkpoint //
		if ( (whatodo == takeshot) && (info > 1) )
		{
			for(ip=0;ip<GPU_N;ip++)
			{
				hipSetDevice(ip);
				storecheckpoints<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
						(
							1,check,nxx,nzz,
							singpu[ip].d_p,singpu[ip].d_vx,singpu[ip].d_vz,
							singpu[ip].d_fipx,singpu[ip].d_fipz,
							singpu[ip].d_fivxx,singpu[ip].d_fivzz,
							singpu[ip].store_p,singpu[ip].store_vx,singpu[ip].store_vz,
							singpu[ip].store_fipx,singpu[ip].store_fipz,
							singpu[ip].store_fivxx,singpu[ip].store_fivzz
						);
						
				if( capo==0 )
				{
					addsource<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
							(
								nxx,nzz,singpu[ip].xsg,singpu[ip].zsg,
								singpu[ip].store_p,singpu[ip].d_rik,dt
							);
				}
						
			}
			
		} // end case 1
		
		
		// advance source wavefield //
		if ( (whatodo == advance) && (info > 2) )
		{
			te = capo;
			for(t=ts;t<=te;t++)
			{	
				for(ip=0;ip<GPU_N;ip++)
				{
					hipSetDevice(ip);
					
					fuzhi_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
								(
									nxx,nzz,singpu[ip].d_vx,singpu[ip].d_vz,singpu[ip].d_dis0,singpu[ip].d_dis1
								);
								
					
				  	hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis0,HIPFFT_FORWARD);
				  	hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis1,singpu[ip].d_dis1,HIPFFT_FORWARD);
				  	
				  	
				  	shift_vxz_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				  				(
				  					singpu[ip].d_kxsmr,singpu[ip].d_kxsmi,
				  					singpu[ip].d_kzscr,singpu[ip].d_kzsci,
				  					singpu[ip].d_dis0,singpu[ip].d_dis1,
				  				        singpu[ip].d_dis2,singpu[ip].d_dis3,
				  				        nxx,nzz
				  				);
				      
				        hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis2,singpu[ip].d_dis2,HIPFFT_BACKWARD); // ps to x
					hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis3,singpu[ip].d_dis3,HIPFFT_BACKWARD); // ps to z

					update_vxx_vzz<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				      				(
				      					nxx,nzz,singpu[ip].d_fivxx,singpu[ip].d_fivzz,
				                                     	singpu[ip].d_i_ax,singpu[ip].d_i_bx,
				                                     	singpu[ip].d_i_az,singpu[ip].d_i_bz,
				                                     	singpu[ip].d_dis2,singpu[ip].d_dis3
				                                ); 

					sumpxpz<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
								(
				      					nxx,nzz,
				                                     	singpu[ip].d_dis2,singpu[ip].d_dis3,singpu[ip].d_dis0
				                                ); 

					hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis1,HIPFFT_FORWARD);

					
					
					multtt_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>(nxx,nzz,singpu[ip].d_k2d1,
											        singpu[ip].d_dis1,singpu[ip].d_dis3);


					hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis3,singpu[ip].d_dis3,HIPFFT_BACKWARD); 


				 	// add dispersion and attenuation terms: inside computational domain //
				  	addsa_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				  				(
				  					restart,pml,nxx,nzz,
				  					singpu[ip].d_vp,singpu[ip].d_rho,dt,
				  					singpu[ip].d_p,singpu[ip].d_dis0,singpu[ip].d_dis3,
				  					singpu[ip].xsg,singpu[ip].zsg,
				  					t,singpu[ip].d_rik,singpu[ip].d_ssg
				  				);

					//hipMemcpy(singpu[ip].h_p,singpu[ip].d_p,NXZ*sizeof(float),hipMemcpyDeviceToHost);

				  	//fwrite(singpu[ip].h_p,sizeof(float),NXZ,fs);

				  	
				  	fuzhi_kernel_p<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
								(
									nxx,nzz,singpu[ip].d_p,singpu[ip].d_dis0
								);
								
					hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis0,HIPFFT_FORWARD);
					
					shift_pxz_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
								(
									singpu[ip].d_kxscr,singpu[ip].d_kxsci,
									singpu[ip].d_kzsmr,singpu[ip].d_kzsmi,
				                                        singpu[ip].d_dis0,
				                                        singpu[ip].d_dis1,singpu[ip].d_dis2,
				                                        nxx,nzz
				                                );

					hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis1,singpu[ip].d_dis1,HIPFFT_BACKWARD);
					hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis2,singpu[ip].d_dis2,HIPFFT_BACKWARD);
	
				      
				      
				      vx_cal_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				      				(
				      					nxx,nzz,dt,singpu[ip].d_bbx,
				      					singpu[ip].d_fipx,singpu[ip].d_h_ax,singpu[ip].d_h_bx,
				      					singpu[ip].d_dis1,singpu[ip].d_vx
				      				);
				      				
				      vz_cal_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				      				(
				      					nxx,nzz,dt,singpu[ip].d_bbz,
				      					singpu[ip].d_fipz,singpu[ip].d_h_az,singpu[ip].d_h_bz,
				      					singpu[ip].d_dis2,singpu[ip].d_vz
				      				);
				} // end gpu
				
			}// end time
			ts = te +1 ;
		}// end case 2
		
		// ----- indicating already finishing forward modeling ----- //
		if ( (whatodo == firsturn) && (info > 2) )
		{
			restart = 1;
			// compute receiver wavefield at t=tmax-1//
			for(ip=0;ip<GPU_N;ip++)
			{
				hipSetDevice(ip);
				hipMemcpyAsync(singpu[ip].d_obs,singpu[ip].h_obs,nrmx*tmax*sizeof(float),hipMemcpyHostToDevice,plansm[ip].stream);
				
				        
				
				fuzhi_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
						(
							nxx,nzz,singpu[ip].d_vxb,singpu[ip].d_vzb,singpu[ip].d_dis0,singpu[ip].d_dis1
						);
						
			
		          	hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis0,HIPFFT_FORWARD);
		          	hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis1,singpu[ip].d_dis1,HIPFFT_FORWARD);
		          	
		          	
		          	shift_vxz_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		          				(
		          					singpu[ip].d_kxsmr,singpu[ip].d_kxsmi,
		          					singpu[ip].d_kzscr,singpu[ip].d_kzsci,
		          					singpu[ip].d_dis0,singpu[ip].d_dis1,
		          				        singpu[ip].d_dis2,singpu[ip].d_dis3,
		          				        nxx,nzz
		          				);
		          	
		          
		          	hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis2,singpu[ip].d_dis2,HIPFFT_BACKWARD); // ps to x
				hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis3,singpu[ip].d_dis3,HIPFFT_BACKWARD); // ps to z

				update_vxx_vzz<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				      				(
				      					nxx,nzz,singpu[ip].d_fivxxb,singpu[ip].d_fivzzb,
				                                     	singpu[ip].d_i_ax,singpu[ip].d_i_bx,
				                                     	singpu[ip].d_i_az,singpu[ip].d_i_bz,
				                                     	singpu[ip].d_dis2,singpu[ip].d_dis3
				                                ); 

					sumpxpz<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
								(
				      					nxx,nzz,
				                                     	singpu[ip].d_dis2,singpu[ip].d_dis3,singpu[ip].d_dis0
				                                ); 

					hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis1,HIPFFT_FORWARD);

					
					multtt_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>(nxx,nzz,singpu[ip].d_k2d1,
											        singpu[ip].d_dis1,singpu[ip].d_dis3);

					

					
					hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis3,singpu[ip].d_dis3,HIPFFT_BACKWARD); 
					

					




		         	// add dispersion and attenuation terms: inside computational domain // 
		          	addsa_b_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		          				(
		          					pml,nxx,nzz,
		          					singpu[ip].d_vp,
		          					singpu[ip].d_rho,dt,
		          					singpu[ip].d_pb,singpu[ip].d_dis0,singpu[ip].d_dis3,
		          					0,tmax,singpu[ip].d_obs,
		          					singpu[ip].d_rindex,singpu[ip].d_flagrec,singpu[ip].nr
		          				);// please note here //


		          	fuzhi_kernel_p<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
							(
								nxx,nzz,singpu[ip].d_pb,singpu[ip].d_dis0
							);
							
				hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis0,HIPFFT_FORWARD);
				
				shift_pxz_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
							(
								singpu[ip].d_kxscr,singpu[ip].d_kxsci,
								singpu[ip].d_kzsmr,singpu[ip].d_kzsmi,
		                                                singpu[ip].d_dis0,
		                                                singpu[ip].d_dis1,singpu[ip].d_dis2,
		                                                nxx,nzz
		                                        );

				hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis1,singpu[ip].d_dis1,HIPFFT_BACKWARD);
				hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis2,singpu[ip].d_dis2,HIPFFT_BACKWARD);
		                                        
		                                        
		                
		                                        
		                                        
		               
		              
		              
		              vx_cal_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		              				(
		              					nxx,nzz,dt,singpu[ip].d_bbx,
		              					singpu[ip].d_fipxb,singpu[ip].d_h_ax,singpu[ip].d_h_bx,
		              					singpu[ip].d_dis1,singpu[ip].d_vxb
		              				);
		              				
		              vz_cal_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		              				(
		              					nxx,nzz,dt,singpu[ip].d_bbz,
		              					singpu[ip].d_fipzb,singpu[ip].d_h_az,singpu[ip].d_h_bz,
		              					singpu[ip].d_dis2,singpu[ip].d_vzb
		              				);
		              				
		              				
		              rtm_image<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		              				(
		              					nx,nz,nxx,nzz,singpu[ip].d_p,singpu[ip].d_pb,
		              					singpu[ip].d_img,singpu[ip].d_imgf
		              				);
		              				
			}// end gpu 

		}// end case 3
		
		
		// the source wavefields are available at this time //
		if ( (whatodo == youturn) && (info > 2) )
		{
			tea = capo; nt = tea;
			// compute receiver wavefield at t=nt //
			for(ip=0;ip<GPU_N;ip++)
			{
				hipSetDevice(ip);
				
				fuzhi_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
						(
							nxx,nzz,singpu[ip].d_vxb,singpu[ip].d_vzb,singpu[ip].d_dis0,singpu[ip].d_dis1
						);
						
			
		          	hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis0,HIPFFT_FORWARD);
		          	hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis1,singpu[ip].d_dis1,HIPFFT_FORWARD);
		          	
		          	
		          	shift_vxz_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		          				(
		          					singpu[ip].d_kxsmr,singpu[ip].d_kxsmi,
		          					singpu[ip].d_kzscr,singpu[ip].d_kzsci,
		          					singpu[ip].d_dis0,singpu[ip].d_dis1,
		          				        singpu[ip].d_dis2,singpu[ip].d_dis3,
		          				        nxx,nzz
		          				);
		          	
		          
		          	hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis2,singpu[ip].d_dis2,HIPFFT_BACKWARD); // ps to x
				hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis3,singpu[ip].d_dis3,HIPFFT_BACKWARD); // ps to z

				update_vxx_vzz<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
				      				(
				      					nxx,nzz,singpu[ip].d_fivxxb,singpu[ip].d_fivzzb,
				                                     	singpu[ip].d_i_ax,singpu[ip].d_i_bx,
				                                     	singpu[ip].d_i_az,singpu[ip].d_i_bz,
				                                     	singpu[ip].d_dis2,singpu[ip].d_dis3
				                                ); 

					sumpxpz<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
								(
				      					nxx,nzz,
				                                     	singpu[ip].d_dis2,singpu[ip].d_dis3,singpu[ip].d_dis0
				                                ); 

					hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis1,HIPFFT_FORWARD);

					
					
					multtt_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>(nxx,nzz,singpu[ip].d_k2d1,
											        singpu[ip].d_dis1,singpu[ip].d_dis3);

					

					
					hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis3,singpu[ip].d_dis3,HIPFFT_BACKWARD); 
					

					




		         	// add dispersion and attenuation terms: inside computational domain // 
		          	addsa_b_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		          				(
		          					pml,nxx,nzz,
		          					singpu[ip].d_vp,
		          					singpu[ip].d_rho,dt,
		          					singpu[ip].d_pb,singpu[ip].d_dis0,singpu[ip].d_dis3,
		          					tmax-1-nt,tmax,singpu[ip].d_obs,
		          					singpu[ip].d_rindex,singpu[ip].d_flagrec,singpu[ip].nr
		          				);// please note here //


		         	

		          	fuzhi_kernel_p<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
							(
								nxx,nzz,singpu[ip].d_pb,singpu[ip].d_dis0
							);
							
				hipfftExecC2C(singpu[ip].plan2dforward,singpu[ip].d_dis0,singpu[ip].d_dis0,HIPFFT_FORWARD);
				
				shift_pxz_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
							(
								singpu[ip].d_kxscr,singpu[ip].d_kxsci,
								singpu[ip].d_kzsmr,singpu[ip].d_kzsmi,
		                                                singpu[ip].d_dis0,
		                                                singpu[ip].d_dis1,singpu[ip].d_dis2,
		                                                nxx,nzz
		                                        );

				hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis1,singpu[ip].d_dis1,HIPFFT_BACKWARD);
				hipfftExecC2C(singpu[ip].plan2dinverse,singpu[ip].d_dis2,singpu[ip].d_dis2,HIPFFT_BACKWARD);
		                                        
		                                        
		                
		                                        
		                                        
		               
		              
		              
		              vx_cal_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		              				(
		              					nxx,nzz,dt,singpu[ip].d_bbx,
		              					singpu[ip].d_fipxb,singpu[ip].d_h_ax,singpu[ip].d_h_bx,
		              					singpu[ip].d_dis1,singpu[ip].d_vxb
		              				);
		              				
		              vz_cal_kernel<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		              				(
		              					nxx,nzz,dt,singpu[ip].d_bbz,
		              					singpu[ip].d_fipzb,singpu[ip].d_h_az,singpu[ip].d_h_bz,
		              					singpu[ip].d_dis2,singpu[ip].d_vzb
		              				);
		              				
		              				
		              rtm_image<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
		              				(
		              					nx,nz,nxx,nzz,singpu[ip].d_p,singpu[ip].d_pb,
		              					singpu[ip].d_img,singpu[ip].d_imgf
		              				);
		              				
				
			}// end gpu iteration
			
		} // end case 4
		
		
		// load stored snapshots //
		if ((whatodo == restore) && (info > 2)) 
		{
			for(ip=0;ip<GPU_N;ip++)
			{
				hipSetDevice(ip);
				
				storecheckpoints<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
						(
							0,check,nxx,nzz,
							singpu[ip].d_p,singpu[ip].d_vx,singpu[ip].d_vz,
							singpu[ip].d_fipx,singpu[ip].d_fipz,
							singpu[ip].d_fivxx,singpu[ip].d_fivzz,
							singpu[ip].store_p,singpu[ip].store_vx,singpu[ip].store_vz,
							singpu[ip].store_fipx,singpu[ip].store_fipz,
							singpu[ip].store_fivxx,singpu[ip].store_fivzz
						);
				
			}
			ts = capo + 1;
		}// end case 5
		
		
		// return error information //
		if ( whatodo == error )
		{
			printf(" irregular termination of revolve \n");
		      	switch(info)
		       	{
				case 10: printf(" number of checkpoints stored exceeds checkup, \n");
				 	 printf(" increase constant 'checkup' and recompile \n");
				 	 break;
				case 11: printf(" number of checkpoints stored = %d exceeds snaps = %d, \n"
						 ,check+1,snaps);
					 printf(" ensure 'snaps' > 0 and increase initial 'fine' \n");
					 break;
				case 12: printf(" error occurs in numforw \n");
					 break;
				case 13: printf(" enhancement of 'fine', 'snaps' checkpoints stored, \n");
					 printf(" increase 'snaps'\n");
					 break;
				case 14: printf(" number of snaps exceeds snapsup, ");
					 printf(" increase constant 'snapsup' and recompile \n");
					 break;
				case 15: printf(" number of reps exceeds repsup, ");
					 printf(" increase constant 'repsup' and recompile \n");       
			}
		} // end case 6
		
		
	}while( (whatodo != terminate) && (whatodo != error) );

	
	
	for(ip=0;ip<GPU_N;ip++)
	{
		hipSetDevice(ip);
		
		filter_image<<<dimGrid,dimBlock,0,plansm[ip].stream>>>
                      				(
                      					nx,nz,dx,dz,singpu[ip].d_img,singpu[ip].d_imgf
                      				);
                      				
                hipMemcpyAsync(singpu[ip].h_img,singpu[ip].d_imgf,nx*nz*sizeof(float),hipMemcpyDeviceToHost,plansm[ip].stream);
                
	        hipMemcpyAsync(singpu[ip].h_ssg,singpu[ip].d_ssg, nx*nz*sizeof(float),hipMemcpyDeviceToHost,plansm[ip].stream);
	        
	        hipStreamSynchronize(plansm[ip].stream);
		
	}
	
	for(ip=0;ip<GPU_N;ip++)
	{
			singpu[ip].lgest=singpu[ip].lgest-pml;
			singpu[ip].rgest=singpu[ip].rgest-pml;

			for(j=0;j<nz;j++)
			{
				for(i=0;i<singpu[ip].lgest;i++){singpu[ip].h_img[i*nz+j]=0.0;}
				for(i=singpu[ip].rgest;i<nx;i++){singpu[ip].h_img[i*nz+j]=0.0;}
			}

			projectomodel(pml,singpu[ip].mlx,dx,nxt,nzt,nx,nz,nxx,nzz,singpu[ip].h_img_pj,singpu[ip].h_img);
			projectomodel(pml,singpu[ip].mlx,dx,nxt,nzt,nx,nz,nxx,nzz,singpu[ip].h_ssg_pj,singpu[ip].h_ssg);

			for(i=0;i<nxt*nzt;i++)
                 	{
				img_sp[i]=img_sp[i]+singpu[ip].h_img_pj[i];
				ssg_sp[i]=ssg_sp[i]+singpu[ip].h_ssg_pj[i];
		 	}
	}
	

    } // end shot iteration


    // free the device and host memory //
    for(i=0;i<GPU_N;i++)
    {
		hipSetDevice(i);

		hipHostFree(singpu[i].h_p);
		hipHostFree(singpu[i].h_img);
		hipHostFree(singpu[i].h_ssg);

		hipfftDestroy(singpu[i].plan2dforward);
		hipfftDestroy(singpu[i].plan2dinverse);

		hipFree(singpu[i].d_p);
		hipFree(singpu[i].d_vx);hipFree(singpu[i].d_vz);
		hipFree(singpu[i].d_fivxx);hipFree(singpu[i].d_fivzz);
		hipFree(singpu[i].d_fipx);hipFree(singpu[i].d_fipz);
		
		hipFree(singpu[i].d_vp);

		hipFree(singpu[i].d_rho);
		hipFree(singpu[i].d_bbx);hipFree(singpu[i].d_bbz);
		
		hipFree(singpu[i].d_rik);

		hipFree(singpu[i].d_obs);
		hipFree(singpu[i].d_img);
		hipFree(singpu[i].d_ssg);
		hipFree(singpu[i].d_imgf);
		hipFree(singpu[i].d_k2d1);

		
		
		hipFree(singpu[i].d_i_ax);hipFree(singpu[i].d_i_bx);
		hipFree(singpu[i].d_i_az);hipFree(singpu[i].d_i_bz);
		hipFree(singpu[i].d_h_ax);hipFree(singpu[i].d_h_bx);
		hipFree(singpu[i].d_h_az);hipFree(singpu[i].d_h_bz);
		
		hipFree(singpu[i].d_dis0);hipFree(singpu[i].d_dis1);
		hipFree(singpu[i].d_dis2);hipFree(singpu[i].d_dis3);hipFree(singpu[i].d_dis4);
		
		
		hipFree(singpu[i].d_flagrec);hipFree(singpu[i].d_rindex);



		// free stored variables //
		hipFree(singpu[i].d_pb);hipFree(singpu[i].d_vxb);hipFree(singpu[i].d_vzb);
		hipFree(singpu[i].d_fivxxb);hipFree(singpu[i].d_fivzzb);
		hipFree(singpu[i].d_fipxb);hipFree(singpu[i].d_fipzb);
		
		hipFree(singpu[i].store_p);
		hipFree(singpu[i].store_vx);hipFree(singpu[i].store_vz);
		hipFree(singpu[i].store_fipx);hipFree(singpu[i].store_fipz);
		hipFree(singpu[i].store_fivxx);hipFree(singpu[i].store_fivzz);
     }
	

     // destroy the multi-streams //
     for(i=0;i<GPU_N;i++)
     {
	 hipSetDevice(i);	
	 hipStreamDestroy(plansm[i].stream);
     }

    return;
	                        	                        
}// end function




void Alloc_host_device_mem(
				int num,int nrmx,int nx,int nz,
				int nxx,int nzz,int nxh,int nzh,int NXZ,int Nh,
			   	int tmax,int GPU_N,struct MultiGPU singpu[]
			  )
{
	int i;
	for(i=0;i<GPU_N;i++)
	{
		hipSetDevice(i);

		hipMalloc((void**)&singpu[i].d_p,sizeof(float)*NXZ);
		hipMalloc((void**)&singpu[i].d_vx,sizeof(float)*NXZ);
		hipMalloc((void**)&singpu[i].d_vz,sizeof(float)*NXZ);
		
		//hipMalloc((void**)&singpu[i].d_aux,sizeof(float)*NXZ);
		//hipMalloc((void**)&singpu[i].d_total,sizeof(float)*NXZ);
    		
    		hipMalloc((void**)&singpu[i].d_fipx,sizeof(float)*NXZ);
    		hipMalloc((void**)&singpu[i].d_fipz,sizeof(float)*NXZ);
    		
    		hipMalloc((void**)&singpu[i].d_fivxx,sizeof(float)*NXZ);
    		hipMalloc((void**)&singpu[i].d_fivzz,sizeof(float)*NXZ);
    
		hipMalloc((void**)&singpu[i].d_vp,sizeof(float)*NXZ);
    		
    		
    		hipMalloc((void**)&singpu[i].d_rho,sizeof(float)*NXZ);
    		hipMalloc((void**)&singpu[i].d_bbx,sizeof(float)*NXZ);
    		hipMalloc((void**)&singpu[i].d_bbz,sizeof(float)*NXZ);
 
    		hipMalloc((void**)&singpu[i].d_rik,sizeof(float)*tmax);
    		//hipMalloc((void**)&singpu[i].d_k2da,sizeof(float)*NXZ);
		hipMalloc((void**)&singpu[i].d_k2d1,sizeof(float)*NXZ);
		//hipMalloc((void**)&singpu[i].d_k2d2,sizeof(float)*NXZ);
		//hipMalloc((void**)&singpu[i].d_k2d3,sizeof(float)*NXZ);
		    
		hipMalloc((void**)&singpu[i].d_i_ax,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_i_bx,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_i_az,sizeof(float)*nzz);
		hipMalloc((void**)&singpu[i].d_i_bz,sizeof(float)*nzz);
		
		hipMalloc((void**)&singpu[i].d_h_ax,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_h_bx,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_h_az,sizeof(float)*nzz);
		hipMalloc((void**)&singpu[i].d_h_bz,sizeof(float)*nzz);
		
		hipMalloc((void**)&singpu[i].d_kxsmr,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_kxscr,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_kxsmi,sizeof(float)*nxx);
		hipMalloc((void**)&singpu[i].d_kxsci,sizeof(float)*nxx);
		
		hipMalloc((void**)&singpu[i].d_kzsmr,sizeof(float)*nzz);
		hipMalloc((void**)&singpu[i].d_kzscr,sizeof(float)*nzz);
		hipMalloc((void**)&singpu[i].d_kzsmi,sizeof(float)*nzz);
		hipMalloc((void**)&singpu[i].d_kzsci,sizeof(float)*nzz);

		hipMalloc((void**)&singpu[i].d_dis0,sizeof(hipfftComplex)*NXZ);
		hipMalloc((void**)&singpu[i].d_dis1,sizeof(hipfftComplex)*NXZ);
		hipMalloc((void**)&singpu[i].d_dis2,sizeof(hipfftComplex)*NXZ);
		hipMalloc((void**)&singpu[i].d_dis3,sizeof(hipfftComplex)*NXZ);
		hipMalloc((void**)&singpu[i].d_dis4,sizeof(hipfftComplex)*NXZ);


	       //host device memory //
               hipHostMalloc((void **)&singpu[i].h_p, sizeof(float)*NXZ);
               hipHostMalloc((void **)&singpu[i].h_img, sizeof(float)*nx*nz);
               hipHostMalloc((void **)&singpu[i].h_ssg, sizeof(float)*nx*nz);
               

	       hipfftPlan2d(&singpu[i].plan2dforward,nxx,nzz,HIPFFT_C2C);
	       hipfftPlan2d(&singpu[i].plan2dinverse,nxx,nzz,HIPFFT_C2C);
	       
	       
	       // for summing gradient //
	       hipMalloc((void**)&singpu[i].d_img,sizeof(float)*nx*nz);
	       hipMalloc((void**)&singpu[i].d_imgf,sizeof(float)*nx*nz);
	       
	       hipMalloc((void**)&singpu[i].d_ssg,sizeof(float)*nx*nz);
	       hipMalloc((void**)&singpu[i].d_obs,sizeof(float)*nrmx*tmax);
	       
	       hipMalloc((void**)&singpu[i].d_flagrec,sizeof(int)*NXZ);
	       hipMalloc((void**)&singpu[i].d_rindex, sizeof(int)*NXZ);
	       
	       
	        // for wavefield reconstruction //
	       hipMalloc((void**)&singpu[i].d_pb,sizeof(hipfftComplex)*NXZ);
	       hipMalloc((void**)&singpu[i].d_vxb,sizeof(hipfftComplex)*NXZ);
	       hipMalloc((void**)&singpu[i].d_vzb,sizeof(hipfftComplex)*NXZ);
	      // hipMalloc((void**)&singpu[i].d_auxb,sizeof(hipfftComplex)*NXZ);
	       
	       hipMalloc((void**)&singpu[i].d_fipxb,sizeof(float)*NXZ);
    	       hipMalloc((void**)&singpu[i].d_fipzb,sizeof(float)*NXZ);
    	       hipMalloc((void**)&singpu[i].d_fivxxb,sizeof(float)*NXZ);
    	       hipMalloc((void**)&singpu[i].d_fivzzb,sizeof(float)*NXZ);
    	       
    	       hipMalloc((void**)&singpu[i].store_p,sizeof(float)*NXZ*num);
    	       hipMalloc((void**)&singpu[i].store_vx,sizeof(float)*NXZ*num);
    	       hipMalloc((void**)&singpu[i].store_vz,sizeof(float)*NXZ*num);
    	       
    	       hipMalloc((void**)&singpu[i].store_fipx,sizeof(float)*NXZ*num);
    	       hipMalloc((void**)&singpu[i].store_fipz,sizeof(float)*NXZ*num);
    	       hipMalloc((void**)&singpu[i].store_fivxx,sizeof(float)*NXZ*num);
    	       hipMalloc((void**)&singpu[i].store_fivzz,sizeof(float)*NXZ*num);
    	      // hipMalloc((void**)&singpu[i].store_aux,sizeof(float)*NXZ*num);
	       

	}
	
	return;
	
}



__global__ void initialize_kernel(
					int nxx,int nzz,int pml,
					float *d_p,float *d_vx,float *d_vz,
					float *d_fipx,float *d_fipz,
					float *d_fivxx,float *d_fivzz,
					float *d_img,float *d_ssg,
					float *d_pb,float *d_vxb,float *d_vzb,
					float *d_fipxb,float *d_fipzb,
					float *d_fivxxb,float *d_fivzzb
				)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       int is=ix*nzz+iz;
       
       int nx=nxx-2*pml;
       int nz=nzz-2*pml;

       if(iz<nzz&&ix<nxx)
       {
           d_p[is]=0.0;
           d_vx[is]=0.0;
           d_vz[is]=0.0;
           d_fipx[is]=0.0;
           d_fipz[is]=0.0;
           d_fivxx[is]=0.0;
           d_fivzz[is]=0.0;
          
           
           d_pb[is]=0.0;
           d_vxb[is]=0.0;
           d_vzb[is]=0.0;
           d_fipxb[is]=0.0;
           d_fipzb[is]=0.0;
           d_fivxxb[is]=0.0;
           d_fivzzb[is]=0.0;
       }
       
       if( iz<nz&&ix<nx&&ix>=0&&iz>=0 )
       {
       		d_img[ix*nz+iz]=0.0;
       		d_ssg[ix*nz+iz]=0.0;
       }

       __syncthreads();
          
}


__global__ void initialize_kernel_ins(
					int nxx,int nzz,int pml,
					float *d_p,float *d_vx,float *d_vz,
					float *d_fipx,float *d_fipz,
					float *d_fivxx,float *d_fivzz,float *d_aux
				)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       int is=ix*nzz+iz;

       if(iz<nzz&&ix<nxx)
       {
           d_p[is]=0.0;
           d_vx[is]=0.0;
           d_vz[is]=0.0;
           d_fipx[is]=0.0;
           d_fipz[is]=0.0;
           d_fivxx[is]=0.0;
           d_fivzz[is]=0.0;
           
           d_aux[is]=0.0;
       }
       __syncthreads();  
}


__global__ void fuzhi_kernel(int nxx,int nzz,float *d_vx,float *d_vz, hipfftComplex *d_dis0, hipfftComplex *d_dis1)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       
       if(iz<nzz&&ix<nxx)
       {
       	  d_dis0[area1].x=d_vx[area1];
          d_dis0[area1].y=0.0;
       	  
          d_dis1[area1].x=d_vz[area1];
          d_dis1[area1].y=0.0;
       }
       __syncthreads();     
}


__global__ void fuzhi_kernel_p (int nxx,int nzz,float *d_p,hipfftComplex *d_dis0)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       if(iz<nzz&&ix<nxx)
       {
       	  d_dis0[area1].x=d_p[area1];
          d_dis0[area1].y=0.0;
       }
       __syncthreads();     
}


__global__ void shift_vxz_kernel(
					float *kxsmr,float *kxsmi,float *kzscr,float *kzsci,
     					hipfftComplex *d_outdatax,hipfftComplex *d_outdataz,
     					hipfftComplex *d_indatax,hipfftComplex *d_indataz,
     					int nxx,int nzz
     				)
{    
     int bx=blockIdx.x;
     int by=blockIdx.y;
     
     int tx=threadIdx.x;
     int ty=threadIdx.y;
     
     int iz=by*BLOCK_HEIGHT+ty;
     int ix=bx*BLOCK_WIDTH+tx;
     
     //int area1=iz*nxx+ix;
     int area2=ix*nzz+iz;
     
     if(iz<nzz&&ix<nxx)
     {   
         if(ix<nxx/2+1)
         {
         
             d_indatax[area2].x=-kxsmr[ix]*d_outdatax[area2].y
                                    -kxsmi[ix]*d_outdatax[area2].x;
             d_indatax[area2].y=kxsmr[ix]*d_outdatax[area2].x
                                   -kxsmi[ix]*d_outdatax[area2].y;
         }
         if(ix>nxx/2)
         {   
             d_indatax[area2].x=kxsmr[ix]*d_outdatax[area2].y
                                   +kxsmi[ix]*d_outdatax[area2].x;
                                   
             d_indatax[area2].y=-kxsmr[ix]*d_outdatax[area2].x
                                    +kxsmi[ix]*d_outdatax[area2].y;
         }
         d_indatax[area2].x=d_indatax[area2].x/nxx/nzz;
         d_indatax[area2].y=d_indatax[area2].y/nxx/nzz;
         
         
         if(iz<nzz/2+1)
         {
             d_indataz[area2].x=-kzscr[iz]*d_outdataz[area2].y
                                    -kzsci[iz]*d_outdataz[area2].x;
             d_indataz[area2].y=kzscr[iz]*d_outdataz[area2].x
                                   -kzsci[iz]*d_outdataz[area2].y;
         }
         if(iz>nzz/2)
         {
             d_indataz[area2].x=kzscr[iz]*d_outdataz[area2].y
                                   +kzsci[iz]*d_outdataz[area2].x;
             d_indataz[area2].y=-kzscr[iz]*d_outdataz[area2].x
                                    +kzsci[iz]*d_outdataz[area2].y;
         }

         d_indataz[area2].x=d_indataz[area2].x/nxx/nzz;
         d_indataz[area2].y=d_indataz[area2].y/nxx/nzz;
     }
     __syncthreads();
}


__global__ void update_vxx_vzz(
       				int nxx,int nzz,float *d_fi_vx,float *d_fi_vz,
       				float *di_a_x,float *di_b_x,float *di_a_z,float *di_b_z,
       				hipfftComplex *d_outdatax,hipfftComplex *d_outdataz
       			       )
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
       float sumx,sumz;
       
       int area2=ix*nzz+iz;
       int area1=area2;
       
       if(iz<nzz&&ix<nxx)
       {
           // calculate fi_vx
           sumx=d_outdatax[area2].x;        
           d_fi_vx[area1]=di_b_x[ix]*d_fi_vx[area1]+di_a_x[ix]*sumx;
           d_outdatax[area2].x=d_outdatax[area2].x+d_fi_vx[area1];
           
           //calculate fi_vz
           sumz=d_outdataz[area1].x;        
           d_fi_vz[area1]=di_b_z[iz]*d_fi_vz[area1]+di_a_z[iz]*sumz;
           d_outdataz[area1].x=d_outdataz[area1].x+d_fi_vz[area1];
       }
       __syncthreads();
}


__global__ void shift_pxz_kernel(
					float *kxscr,float *kxsci,float *kzsmr,float *kzsmi,
     					hipfftComplex *d_dis0,
     					hipfftComplex *d_dis1,hipfftComplex *d_dis2,
     					int nxx,int nzz
     				)
{   
     int bx=blockIdx.x;
     int by=blockIdx.y;
     
     int tx=threadIdx.x;
     int ty=threadIdx.y;
     
     int iz=by*BLOCK_HEIGHT+ty;
     int ix=bx*BLOCK_WIDTH+tx;
     
     int area2=ix*nzz+iz;
     int area1=area2;
     
     if(iz<nzz&&ix<nxx)
     {   
         if(ix<nxx/2+1)
         {
         
             d_dis1[area1].x=-kxscr[ix]*d_dis0[area1].y
                                    -kxsci[ix]*d_dis0[area1].x;
             d_dis1[area1].y=kxscr[ix]*d_dis0[area1].x
                                   -kxsci[ix]*d_dis0[area1].y;
         }
         if(ix>nxx/2)
         {   
             d_dis1[area1].x=kxscr[ix]*d_dis0[area1].y
                                   +kxsci[ix]*d_dis0[area1].x;
                                   
             d_dis1[area1].y=-kxscr[ix]*d_dis0[area1].x
                                    +kxsci[ix]*d_dis0[area1].y;
         }
         d_dis1[area1].x=d_dis1[area1].x/nxx/nzz;
         d_dis1[area1].y=d_dis1[area1].y/nxx/nzz;
         
         
         if(iz<nzz/2+1)
         {
             d_dis2[area2].x=-kzsmr[iz]*d_dis0[area2].y
                                    -kzsmi[iz]*d_dis0[area2].x;
             d_dis2[area2].y=kzsmr[iz]*d_dis0[area2].x
                                   -kzsmi[iz]*d_dis0[area2].y;
         }
         if(iz>nzz/2)
         {
             d_dis2[area2].x=kzsmr[iz]*d_dis0[area2].y
                                   +kzsmi[iz]*d_dis0[area2].x;
             d_dis2[area2].y=-kzsmr[iz]*d_dis0[area2].x
                                    +kzsmi[iz]*d_dis0[area2].y;
         }

         d_dis2[area2].x=d_dis2[area2].x/nxx/nzz;
         d_dis2[area2].y=d_dis2[area2].y/nxx/nzz;
     }
     __syncthreads();
} 


__global__ void vx_cal_kernel(
     				int nxx,int nzz,float dt,float *d_bbx,
     				float *d_fi_px,float *dh_a_x,
     				float *dh_b_x,hipfftComplex *d_outdatax,float *d_vx)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
       float sumx;
       
       int area2=ix*nzz+iz;
       int area1=area2;
       
       if(iz<nzz&&ix<nxx)
       {    
           // calculate fi_px
           sumx=d_outdatax[area1].x;
           d_fi_px[area2]=dh_b_x[ix]*d_fi_px[area2]+dh_a_x[ix]*sumx;
           
           // calculate vx
           d_vx[area2]=d_vx[area2]+dt*d_bbx[area1]*(d_outdatax[area1].x+d_fi_px[area2]);
       }
       __syncthreads();   
}

__global__ void vz_cal_kernel(
				int nxx,int nzz,float dt,float *d_bbz,
				float *d_fi_pz,float *dh_a_z,
    				float *dh_b_z,hipfftComplex *d_outdataz,float *d_vz)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
       float sumz;
       
       int area1=ix*nzz+iz;
       
       if(iz<nzz&&ix<nxx)
       {    
           // calculate fi_pz
           sumz=d_outdataz[area1].x;
           d_fi_pz[area1]=dh_b_z[iz]*d_fi_pz[area1]+dh_a_z[iz]*sumz;
           
           // calculate vz
           d_vz[area1]=d_vz[area1]+dt*d_bbz[area1]*(d_outdataz[area1].x+d_fi_pz[area1]);
       }
       __syncthreads();   
}



__global__ void multv_kernel(int N1,int nxx,int nzz,int ig,float *d_V,hipfftComplex *d_psk,hipfftComplex *d_pskf)
{        
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       int NXZ=N1;
       int area2=ig*NXZ+area1;

       if(iz<nzz&&ix<nxx)
       {
          d_pskf[area1].x=d_psk[area1].x*d_V[area2]/NXZ;
          d_pskf[area1].y=d_psk[area1].y*d_V[area2]/NXZ;

       }
       __syncthreads();     
}


__global__ void fuzhi_to_ifftrow(int ig,int nxx,int nzz,int N1,hipfftComplex *d_pskf,hipfftComplex *d_pifft)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       int area2=ig*N1+area1;

       if(iz<nzz&&ix<nxx)
       {
          d_pifft[area2].x=d_pskf[area1].x;
          d_pifft[area2].y=d_pskf[area1].y;
       }
       __syncthreads();     
}


__global__ void fuzhi_r_kernel(int nxx,int nzz,int jg,int N1,double *d_pst)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       int area2=jg*N1+area1;

       if(iz<nzz&&ix<nxx)
       {
          d_pst[area2]=0.0;
       }
       __syncthreads();     
}

__global__ void sumlw_kernel(int nxx,int nzz,int jg,int N1,int row,int ig,double *d_pst,double *d_A,hipfftComplex *d_pifft)
{

       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       int area2=jg*N1+area1;
       int area3=ig*N1+area1;

       if(iz<nzz&&ix<nxx)
       {
          d_pst[area2]=d_pst[area2]+d_A[jg*row+ig]*d_pifft[area3].x;
       }
       __syncthreads();     

}

__global__ void sumlw2_kernel(int nxx,int nzz,int N1,int col,double *d_pst,float *d_U,float *d_psc)
{

       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       int j,is;
       double temp;

       if(iz<nzz&&ix<nxx)
       {
          temp=0.0;
          for(j=0;j<col;j++)
          {
              is=area1*col+j;
              temp=temp+d_U[is]*d_pst[j*N1+area1];
          }
          d_psc[area1]=temp;
          
       }
       __syncthreads();     

}




__global__ void addsa_kernel(
				int restart,int pml,int nxx,int nzz,
				float *d_vp,float *d_rho,float dt,
				float *d_p,hipfftComplex *d_dis0,hipfftComplex *d_dis3,
				int xs,int zs,int nt,float *d_Rik,float *d_ssg
			    )
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       
       int nx=nxx-2*pml;
       int nz=nzz-2*pml;
       
       int ixx=ix-pml;
       int izz=iz-pml;
       int area2=ixx*nz+izz;
       
       if( iz<nzz&&ix<nxx )
       {
	   d_p[area1]=d_p[area1]+d_rho[area1]*d_vp[area1]*d_vp[area1]*dt*(d_dis0[area1].x-1.0/12.0*dt*d_vp[area1]*dt*d_vp[area1]*d_dis3[area1].x);

          
          
          if(iz==zs&&ix==xs){d_p[area1]=d_p[area1]+d_Rik[nt]*dt*1000.0;}
          
          if( restart==0&& ixx>=0&&ixx<nx&&izz>=0&&izz<nz ){ d_ssg[area2] = d_ssg[area2] + d_p[area1]*d_p[area1]; }
       }
       __syncthreads();     
}


__global__ void addsa_b_kernel(
				int pml,int nxx,int nzz,
				float *d_vp,float *d_rho,float dt,
				float *d_p,hipfftComplex *d_dis0,hipfftComplex *d_dis3,
				int nt,int tmax,float *d_obs,
				int *rindex, int *flagrec,int nr
			    )
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       int area1=ix*nzz+iz;
       
       
       if( iz<nzz && ix<nxx && ix>=0 && iz>=0 )
       {

	  d_p[area1]=d_p[area1]+d_rho[area1]*d_vp[area1]*d_vp[area1]*dt*(d_dis0[area1].x-1.0/12.0*dt*d_vp[area1]*dt*d_vp[area1]*d_dis3[area1].x);
          
          if( flagrec[area1]==1 ){ d_p[area1]= d_obs[ (tmax-1-nt)*nr+rindex[area1] ]; }  
       }
       
       
       __syncthreads();     
	
}

__global__ void rtm_image(
				int nx,int nz,int nxx,int nzz,
				float *d_p,float *d_pb,
				float *d_img,float *d_imgf
			 )
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
       int pml=(nxx-nx)/2;
       
       int area1 = ix*nzz+iz;
       int area2 = (ix-pml)*nz+iz-pml;
       
       if ( ix>=pml&&ix<nxx-pml&&iz>=pml&&iz<nzz-pml )
       {
       		d_img[area2] = d_img[area2] + d_p[area1]*d_pb[area1];
       		
       		d_imgf[area2] = d_img[area2];
       }
	
}
                      				
__global__ void filter_image(int nx,int nz,float dx,float dz,float *d_img,float *d_imgf)
{
	int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
       int area1 = ix*nz+iz;
       
       if ( ix>=1&&ix<nx-1&&iz>=1&&iz<nz-1 )
       {
       		d_imgf[area1] = (d_img[area1+1] + d_img[area1-1] -2.0*d_img[area1])/dz/dz
       				+ (d_img[area1+nz] + d_img[area1-nz]-2.0*d_img[area1])/dx/dx;
       }
       
       if (ix==0||iz==0||ix==nx-1||iz==nz-1){d_imgf[area1]=0.0;}
}


__global__ void multt_kernel(float eta,float dt,int nt,float tao,
			     int NXZ,int nxx,int nzz,
			     float *k2da,hipfftComplex *d_dis0
			     )
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       float tt=nt*dt;
       float tmp;

       if(iz<nzz&&ix<nxx)
       {
       	     
       	  tmp=k2da[area1]-2.0*eta*k2da[area1]*(1.0+tao*dt*k2da[area1])/10000.0/( expf(-tao*tt*k2da[area1]) + eta*expf(tao*dt*k2da[area1])/10000.0 );
       	  //tmp=k2da[area1];
       	  
          d_dis0[area1].x=d_dis0[area1].x*tmp/NXZ;
          d_dis0[area1].y=d_dis0[area1].y*tmp/NXZ;
       }
	
	__syncthreads();
}

__global__ void fuzhi_kernel_inverse(int nxx,int nzz,float *d_aux,hipfftComplex *d_dis0)
{
	int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       if(iz<nzz&&ix<nxx)
       {
       	  d_aux[area1]=d_dis0[area1].x;
       }
       __syncthreads();     
	
}

__global__ void storecheckpoints
		(
			int flag,int nc,int nxx,int nzz,
			float *d_p,float *d_vx,float *d_vz,
			float *d_fipx,float *d_fipz,
			float *d_fivxx,float *d_fivzz,
			float *store_p,float *store_vx,float *store_vz,
			float *store_fipx,float *store_fipz,
			float *store_fivxx,float *store_fivzz
		)
{
	 int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
       int area=ix*nzz+iz;
       int area1=nc*nxx*nzz+area;
       
       if (ix>=0&&ix<nxx&&iz>=0&&iz<nzz)
       {
       		if(flag==1)
       		{
       			store_p[area1]  = d_p[area];
       			store_vx[area1] = d_vx[area];
       			store_vz[area1] = d_vz[area];
       			store_fipx[area1] = d_fipx[area];
       			store_fipz[area1] = d_fipz[area];
       			store_fivxx[area1] = d_fivxx[area];
       			store_fivzz[area1] = d_fivzz[area];
       		}
       		else
       		{
       			d_p[area]  = store_p[area1];
       			d_vx[area] = store_vx[area1];
       			d_vz[area] = store_vz[area1];
       			
       			d_fipx[area]  = store_fipx[area1];
       			d_fipz[area]  = store_fipz[area1];
       			d_fivxx[area] = store_fivxx[area1];
       			d_fivzz[area] = store_fivzz[area1];
       		}
       }
}

__global__ void addsource(int nx,int nz,int xs,int zs,float *check_d_ps,float *d_rik,float dt)
{
	int bx=blockIdx.x;
        int by=blockIdx.y;
     
	int tx=threadIdx.x;
        int ty=threadIdx.y;
     
        int iz=by*BLOCK_HEIGHT+ty;
        int ix=bx*BLOCK_WIDTH+tx;

	int is=ix*nz+iz;

	if(iz==zs&&ix==xs)
	{
		check_d_ps[is]=check_d_ps[is]+d_rik[0]*dt*1000.0;
	}
}

__global__ void sumpxpz(
       				int nxx,int nzz,
       				hipfftComplex *d_outdatax,hipfftComplex *d_outdataz,
				hipfftComplex *d_dis0
       			       )
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
      
       
       int area2=ix*nzz+iz;
       //int area1=area2;
       
       if(iz<nzz&&ix<nxx)
       {
		d_dis0[area2].x=d_outdatax[area2].x+d_outdataz[area2].x;
		d_dis0[area2].y=d_outdatax[area2].y+d_outdataz[area2].y;
		//d_dis0[area2].y=0.0;
       }
       __syncthreads();
}

__global__ void multtt_kernel(int nxx,int nzz,float *d_k2d3,hipfftComplex *d_dis0,hipfftComplex *d_dis1)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;

       int area1=ix*nzz+iz;
       int NXZ=nxx*nzz;

       if(iz<nzz&&ix<nxx)
       {
          d_dis1[area1].x=d_dis0[area1].x*d_k2d3[area1]/NXZ;
          d_dis1[area1].y=d_dis0[area1].y*d_k2d3[area1]/NXZ;

       }
       __syncthreads();     
}

__global__ void  sum_total(int nxx,int nzz,
			   hipfftComplex *d_dis0,hipfftComplex *d_dis2,
			   hipfftComplex *d_dis3,hipfftComplex *d_dis4,
			   float *d_total,float *d_vp,float *d_lam,
			   float *d_gam,float *d_vwd,float dt)
{
       int bx=blockIdx.x;
       int by=blockIdx.y;
     
       int tx=threadIdx.x;
       int ty=threadIdx.y;
     
       int iz=by*BLOCK_HEIGHT+ty;
       int ix=bx*BLOCK_WIDTH+tx;
       
        int area2=ix*nzz+iz;
	float y1,y2,y3,y4;
	float dtf=dt*dt;
	float vpf;

	if(iz<nzz&&ix<nxx)
       {
		y1=1.0-32.0*d_gam[area2];
		y2=32.0*d_gam[area2]*d_vwd[area2];
		y3=1.0-64.0*d_gam[area2];
		y4=64.0*d_gam[area2]*d_vwd[area2];
		vpf=d_vp[area2]*d_vp[area2];
	
	d_total[area2]= y1*d_dis0[area2].x+y2*d_dis2[area2].x-1.0/12.0*dtf*vpf*d_lam[area2]*(y3*d_dis3[area2].x+y4*d_dis4[area2].x);
       }
       __syncthreads();
	
}
